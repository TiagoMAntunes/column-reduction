#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <chrono>

#define CUDA_CHECK(status) (assert(status == hipSuccess))
#define threads_per_block 1024

__global__ void column_reduce(float * matrix, float * result, int m /* lines */, int n /* columns*/) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x; // line
    unsigned int i = threadIdx.x * n + threadIdx.y + blockIdx.y * blockDim.y; // get to idx th line
    unsigned int offset = 0;
    unsigned int it = n * blockDim.x; // advance blockDim.x threads vertically
    unsigned int real_y = blockIdx.y * blockDim.y + threadIdx.y;

    // sum all the values from that column to fit in one single block
    sdata[tid] = 0;
    if (real_y < n && threadIdx.x < m) // remember we only have one x block
        while (i + offset < n*m) {
            sdata[tid] += matrix[i + offset];
            offset += it; 
            
        }
    __syncthreads();

    unsigned int lowest = blockDim.x > m ? m : blockDim.x;
    if (real_y < n && threadIdx.x < m)
        for (unsigned int s = 1; threadIdx.x + s < lowest; s *= 2) {
            if (threadIdx.x % (2*s) == 0) {
                sdata[tid] += sdata[tid + s];
            }

            __syncthreads();
        }

    if (threadIdx.x == 0 && real_y < n) {
        result[real_y] = sdata[tid];
    }

}

int main(int argc, char * argv[])  {
    if (argc < 3) {
        printf("Usage: %s <m> <n>\n", argv[0]);
        return 0;
    }

    int m = atoi(argv[1]), n = atoi(argv[2]);
    
    unsigned long seed = time(NULL);
    srand(seed); // seed 
    printf("Running with seed %ld\n", seed);

    // create row-major matrix m x n
    float * matrix = (float *) malloc(sizeof(float) * m * n); // m x n

    // create array to store result
    float * result_gpu = (float *) malloc(sizeof(float) * n); // tot_num_blocks x 1
    float * result_cpu = (float *) malloc(sizeof(float) * n); // validation
    memset(result_cpu, 0, sizeof(float) * n);

    printf("Populating array \n");
    // populate the array 
    for (int i = 0; i < m * n; i++)
        matrix[i] = 1.0 / ((rand() % 977) + 1);
    

    printf("Calculating final result\n");
    auto cpu_start = std::chrono::high_resolution_clock::now();
    // calculate cpu result
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++) 
            result_cpu[j] += matrix[i * n + j];
    auto cpu_end = std::chrono::high_resolution_clock::now();

    printf("CPU took %f ms.\n", std::chrono::duration_cast<std::chrono::microseconds>(cpu_end - cpu_start).count() / 1000.0);
    

    printf("Allocating GPU memory, m=%d, n=%d\n", m, n);
    // allocate gpu memory
    float * matrix_gpu, * device_result, * helper_result = NULL;
    
    CUDA_CHECK(hipMalloc(&matrix_gpu, sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(&device_result, sizeof(float) * n));
    CUDA_CHECK(hipMemset(device_result, 0, sizeof(float) * n));
    printf("Finished allocating. Copying matrix...\n");
    // move matrix into gpu
    CUDA_CHECK(hipMemcpy(matrix_gpu, matrix, m * n * sizeof(float), hipMemcpyHostToDevice));

    printf("Calling kernel with m=%d n=%d\n", m, n);
    hipEvent_t start, stop;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // call kernel
    dim3 block_threads(32, 32);
    dim3 grid_threads(1, n / 32 + (n % 32 ? 1 : 0));

    CUDA_CHECK(hipEventRecord(start));
    column_reduce<<<grid_threads, block_threads, sizeof(float)*threads_per_block>>>(matrix_gpu, device_result, m, n);
    CUDA_CHECK(hipEventRecord(stop));

    // Wait for final kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());
    // end = std::chrono::high_resolution_clock::now();
    
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Kernel finished. Took %f ms. Copying back results.\n", elapsed_time);
    // copy back results
    CUDA_CHECK(hipMemcpy(result_gpu, device_result, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free gpu memory
    CUDA_CHECK(hipFree(matrix_gpu));
    CUDA_CHECK(hipFree(device_result));
    if (helper_result) CUDA_CHECK(hipFree(helper_result));
    
    printf("Released GPU memory. Validating results...\n");
    // compare results
    for (int i = 0; i < n; i++) {
        if (abs(result_cpu[i] - result_gpu[i]) > 1e-3) 
            printf("INCORRECT RESULT: %.10f %.10f @ %d, diff=%.10f\n", result_cpu[i], result_gpu[i], i, result_cpu[i] - result_gpu[i]);
        // else printf("Correct result! cpu=%.10f, gpu=%.10f, diff=%.10f\n", result_cpu[i], result_gpu[i], result_cpu[i] - result_gpu[i]);
    }
    
    free(result_gpu);
    free(result_cpu);
    free(matrix);
    return 0;
}