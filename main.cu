#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>

#define CUDA_CHECK(status) (assert(status == hipSuccess))
#define threads_per_block 1024

__global__ void column_reduce(float * matrix, float * result, int m /* lines */, int n /* rows*/, int num_blocks_per_line) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int curr_block_limit = (blockIdx.x == (num_blocks_per_line - 1)) ? n % threads_per_block : blockDim.x;
    unsigned int line_block_idx = blockIdx.x % num_blocks_per_line;
    //                        matrix line                  +        previous blocks in line     + id in block
    unsigned int i = blockIdx.x / num_blocks_per_line  * n + line_block_idx * threads_per_block + threadIdx.x;
    
    if (threadIdx.x < curr_block_limit) {
        // if (blockIdx.x != 0) printf("Thread %d Block %d has i=%d %d\n", threadIdx.x, blockIdx.x, i, curr_block_limit);
        if (i < m * n) sdata[tid] = matrix[i];
        __syncthreads();

        for (unsigned int s = 1; s < blockDim.x; s *= 2) {
            if (tid % (2*s) == 0) {
                sdata[tid] += sdata[tid + s];
            }

            __syncthreads();
        }

        if (tid == 0) result[blockIdx.x] = sdata[0];
    }

}

int main(int argc, char * argv[])  {
    if (argc < 3) {
        printf("Usage: %s <m> <n>\n", argv[0]);
        return 0;
    }

    int m = atoi(argv[1]), n = atoi(argv[2]);
    
    int num_blocks_per_line = n / threads_per_block + (n % threads_per_block ? 1 : 0);
    int tot_num_blocks = m * num_blocks_per_line;

    printf("Running with %d threads per block, %d num blocks per line, %d tot number blocks\n", threads_per_block, num_blocks_per_line, tot_num_blocks);

    srand(time(NULL)); // seed 

    // create row-major matrix m x n
    float * matrix = (float *) malloc(sizeof(float) * m * n); // m x n
    // create array to store result
    float * result_gpu = (float *) malloc(sizeof(float) * tot_num_blocks); // tot_num_blocks x 1
    float * result_cpu = (float *) malloc(sizeof(float) * m); // validation

    printf("Populating array \n");
    // populate the array 
    for (int i = 0; i < m * n; i++)
        matrix[i] = 1.0 / ((rand() % 977) + 1);
    

    printf("Calculating final result\n");
    // calculate cpu result
    for (int i = 0; i < m; i++) {
        int row = i * n;
        result_cpu[i] = 0;
        for (int j = 0; j < n; j++) {
            result_cpu[i] += matrix[j + row];
        }
    }

    // printf("--- Result CPU ---\n");
    // for (int i = 0; i < m; i++) {
    //     printf("%0.4f = ", result_cpu[i]);
    //     for (int j = 0; j < n; j++) {
    //         printf("%0.4f ", matrix[j + i * n]);
    //     }
    //     printf("\n");
    // }

    printf("Allocating GPU memory\n");
    // allocate gpu memory
    float * matrix_gpu, * device_result;
    CUDA_CHECK(hipMalloc(&matrix_gpu, sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(&device_result, sizeof(float) * tot_num_blocks));
    
    // move matrix into gpu
    CUDA_CHECK(hipMemcpy(matrix_gpu, matrix, m * n * sizeof(float), hipMemcpyHostToDevice));

    printf("Calling kernel\n");
    // call kernel
    column_reduce<<<tot_num_blocks, threads_per_block, sizeof(float)*threads_per_block>>>(matrix_gpu, device_result, m, n, num_blocks_per_line);

    printf("Kernel launched. Waiting...\n");
    // Wait for kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    printf("Kernel finished. Copying back results.\n");
    // copy back results
    CUDA_CHECK(hipMemcpy(result_gpu, device_result, m * sizeof(float), hipMemcpyDeviceToHost));
    
    // free gpu memory
    CUDA_CHECK(hipFree(matrix_gpu));
    CUDA_CHECK(hipFree(device_result));
    
    printf("Released GPU memory. Validating results...\n");
    // compare results
    for (int i = 0; i < m; i++) {
        if (result_cpu[i] - result_gpu[i] > 1e-4) 
            printf("INCORRECT RESULT: %.10f %.10f @ %d\n", result_cpu[i], result_gpu[i], i);
    }
    
    free(result_gpu);
    free(result_cpu);
    free(matrix);
    return 0;
}