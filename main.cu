#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>

#define CUDA_CHECK(status) (assert(status == hipSuccess))
#define threads_per_block 1024

__global__ void column_reduce(float * matrix, float * result, int m /* lines */, int n /* columns*/) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * n + threadIdx.x; // get to idx th line
    unsigned int offset = 0;

    // sum all the values from that line to fit in one single block
    sdata[tid] = 0;
    while (tid + offset < n) {
        sdata[tid] += matrix[i + offset];
        offset += blockDim.x;
    }
    __syncthreads();

    for (unsigned int s = 1; tid + s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    if (tid == 0) {result[blockIdx.x] = sdata[0];}

}

int main(int argc, char * argv[])  {
    if (argc < 3) {
        printf("Usage: %s <m> <n>\n", argv[0]);
        return 0;
    }

    int m = atoi(argv[1]), n = atoi(argv[2]);
    
    unsigned long seed = time(NULL);
    srand(seed); // seed 
    printf("Running with seed %ld\n", seed);

    // create row-major matrix m x n
    float * matrix = (float *) malloc(sizeof(float) * m * n); // m x n
    // create array to store result
    float * result_gpu = (float *) malloc(sizeof(float) * m); // tot_num_blocks x 1
    float * result_cpu = (float *) malloc(sizeof(float) * m); // validation

    printf("Populating array \n");
    // populate the array 
    for (int i = 0; i < m * n; i++)
        matrix[i] = 1.0 / ((rand() % 977) + 1);
    

    printf("Calculating final result\n");
    // calculate cpu result
    for (int i = 0; i < m; i++) {
        int row = i * n;
        result_cpu[i] = 0;
        for (int j = 0; j < n; j++) {
            result_cpu[i] += matrix[j + row];
        }
    }

    // printf("--- Result CPU ---\n");
    // for (int i = 0; i < m; i++) {
    //     printf("%0.10f = ", result_cpu[i]);
    //     for (int j = 0; j < n; j++) {
    //         printf("%0.10f ", matrix[j + i * n]);
    //     }
    //     printf("\n");
    // }

    printf("Allocating GPU memory\n");
    // allocate gpu memory
    float * matrix_gpu, * device_result, * helper_result = NULL;

    CUDA_CHECK(hipMalloc(&matrix_gpu, sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(&device_result, sizeof(float) * m));
    CUDA_CHECK(hipMemset(device_result, 0, sizeof(float) * m));

    // move matrix into gpu
    CUDA_CHECK(hipMemcpy(matrix_gpu, matrix, m * n * sizeof(float), hipMemcpyHostToDevice));

    // call kernel
    column_reduce<<<m, threads_per_block, sizeof(float)*threads_per_block>>>(matrix_gpu, device_result, m, n);

    // Wait for final kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    printf("Kernel finished. Copying back results.\n");
    // copy back results
    CUDA_CHECK(hipMemcpy(result_gpu, device_result, m * sizeof(float), hipMemcpyDeviceToHost));
    
    // free gpu memory
    CUDA_CHECK(hipFree(matrix_gpu));
    CUDA_CHECK(hipFree(device_result));
    if (helper_result) CUDA_CHECK(hipFree(helper_result));
    
    printf("Released GPU memory. Validating results...\n");
    // compare results
    for (int i = 0; i < m; i++) {
        if (abs(result_cpu[i] - result_gpu[i]) > 1e-4) 
            printf("INCORRECT RESULT: %.10f %.10f @ %d, diff=%.10f\n", result_cpu[i], result_gpu[i], i, result_cpu[i] - result_gpu[i]);
        // else printf("Correct result! cpu=%.10f, gpu=%.10f, diff=%.10f\n", result_cpu[i], result_gpu[i], result_cpu[i] - result_gpu[i]);
    }
    
    free(result_gpu);
    free(result_cpu);
    free(matrix);
    return 0;
}