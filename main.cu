#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>

#define CUDA_CHECK(status) (assert(status == hipSuccess))


__global__ void column_reduce(float * matrix, float * result, int m /* lines */, int n /* rows*/) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m * n) sdata[tid] = matrix[i];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    if (tid == 0) result[blockIdx.x] = sdata[0];

}

int main(int argc, char * argv[])  {
    if (argc < 3) {
        printf("Usage: %s <m> <n>\n", argv[0]);
        return 0;
    }

    int m = atoi(argv[1]), n = atoi(argv[2]);
    
    srand(time(NULL)); // seed 

    // create row-major matrix m x n
    float * matrix = (float *) malloc(sizeof(float) * m * n); // m x n
    // create array to store result
    float * result_gpu = (float *) malloc(sizeof(float) * m); // m x 1
    float * result_cpu = (float *) malloc(sizeof(float) * m); // validation

    printf("Populating array \n");
    // populate the array 
    for (int i = 0; i < m * n; i++)
        matrix[i] = 1.0 / ((rand() % 977) + 1);
    

    printf("Calculating final result\n");
    // calculate cpu result
    for (int i = 0; i < m; i++) {
        int row = i * n;
        result_cpu[i] = 0;
        for (int j = 0; j < n; j++) {
            result_cpu[i] += matrix[j + row];
        }
    }

    // printf("--- Result CPU ---\n");
    // for (int i = 0; i < m; i++) {
    //     printf("%0.4f = ", result_cpu[i]);
    //     for (int j = 0; j < n; j++) {
    //         printf("%0.4f ", matrix[j + i * n]);
    //     }
    //     printf("\n");
    // }

    printf("Allocating GPU memory\n");
    // allocate gpu memory
    float * matrix_gpu, * device_result;
    CUDA_CHECK(hipMalloc(&matrix_gpu, sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(&device_result, sizeof(float) * m));
    
    // move matrix into gpu
    CUDA_CHECK(hipMemcpy(matrix_gpu, matrix, m * n * sizeof(float), hipMemcpyHostToDevice));

    printf("Calling kernel\n");
    // call kernel
    dim3 threadsPerBlock(n); // each block is a row in the matrix
    dim3 numBlocks(m);
    column_reduce<<<m, n, sizeof(float)*n>>>(matrix_gpu, device_result, m, n);

    printf("Kernel launched. Waiting...\n");
    // Wait for kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    printf("Kernel finished. Copying back results.\n");
    // copy back results
    CUDA_CHECK(hipMemcpy(result_gpu, device_result, m * sizeof(float), hipMemcpyDeviceToHost));
    
    // free gpu memory
    CUDA_CHECK(hipFree(matrix_gpu));
    CUDA_CHECK(hipFree(device_result));
    
    printf("Released GPU memory. Validating results...\n");
    // compare results
    for (int i = 0; i < m; i++) {
        if (result_cpu[i] - result_gpu[i] > 1e-4) 
            printf("INCORRECT RESULT: %.10f %.10f @ %d\n", result_cpu[i], result_gpu[i], i);
    }
    
    free(result_gpu);
    free(result_cpu);
    free(matrix);
    return 0;
}